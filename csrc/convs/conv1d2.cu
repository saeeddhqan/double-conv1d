#include "hip/hip_runtime.h"
// #include <hip/hip_runtime.h>
// #include <hip/hip_runtime.h>

// #include <ATen/cuda/HIPContext.h>
// #include <torch/extension.h>
#include "shared.h"

template <typename input_t, typename weight_t, int kNWarpsPerBlock, int kNChunksPerSequence>
__global__ void scan(
    const weight_t* __restrict__ gates,
    const input_t* __restrict__ tokens,
    input_t* __restrict__ out,
    const int batch_stride,
    const int dim_stride,
    const int batch_stride_o
) {
    __shared__ weight_t warpLastToken[kNWarpsPerBlock];
    const uint lane_id = threadIdx.x & 31; // x % 32

    int odd = lane_id & 1;

    weight_t x = tokens[(blockIdx.x * batch_stride + blockIdx.y * dim_stride) + threadIdx.x] * gates[(256 * blockIdx.z) + threadIdx.x];

    #pragma unroll
    for (int delta = 1; delta < 32; delta *= 2) {
        weight_t prev_x = __shfl_up_sync(0xffffffff, x, delta);
        if (lane_id >= delta && odd) {
            x += prev_x;
            odd = (lane_id % (delta * 4)) == (delta * 4) - 1;
        }
    }
    __syncwarp();
    if (lane_id == 31) {
        warpLastToken[(threadIdx.x / 32)] = x;
    }
    __syncthreads();
    if (threadIdx.x % 256 == 255) {
        x += warpLastToken[0];
        x += warpLastToken[1];
        x += warpLastToken[2];
        x += warpLastToken[3];
        x += warpLastToken[4];
        x += warpLastToken[5];
        x += warpLastToken[6];
        out[(blockIdx.x * batch_stride_o) + (blockIdx.y * 64) + blockIdx.z] = x;
    }
}

// template <typename weight_t, typename torch_weight_t>
// void
// warpscan(const at::Tensor &gates, const at::Tensor &tokens, const at::Tensor &out, const bool reverse) {
//     const auto strides = tokens.strides();
//     const int batch_stride = strides[0];
//     const int dim_stride = strides[1];
//     const int gate_stride = gates.size(0) / 8;
//     TORCH_CHECK(tokens.stride(-1) == 1 || tokens.size(-1) == 1);
//     TORCH_CHECK(gates.stride(-1) == 1 || gates.size(-1) == 1);
//     const int batch_stride_o = out.strides()[0];
//     const auto sizes = tokens.sizes();
//     const int batch_size = sizes[0];
//     const int dim = sizes[1];
//     const int seqlen = sizes[2];

//     auto stream = at::cuda::getCurrentCUDAStream().stream();
//     // I guess the number of blocks are high, the performance is poor
//     dim3 grid(batch_size, dim, out.strides()[1]); // 4 is sequential steps;

//     if (seqlen == 256) {
//         constexpr int kNWarpsPerBlock = 8;
//         constexpr int kNChunksPerSequence = 1;
//         scan<weight_t, kNWarpsPerBlock, kNChunksPerSequence><<<grid, seqlen>>>(
//             reinterpret_cast<weight_t*>(gates.data_ptr<torch_weight_t>()), reinterpret_cast<weight_t*>(tokens.data_ptr<torch_weight_t>()), reinterpret_cast<weight_t*>(out.data_ptr<torch_weight_t>()),
//             batch_stride, dim_stride, batch_stride_o
//         );
//     } else {
//         TORCH_CHECK(false && "seqlen must be a power of 2, >= 32, <= 65536");
//     }
// }

torch::Tensor warpscan(const torch::Tensor gates, const torch::Tensor tokens) {
    const auto strides = tokens.strides();
    const int batch_stride = strides[0];
    const int dim_stride = strides[1];

    const int B = tokens.size(0);
    const int D = tokens.size(1);
    const int seqlen = tokens.size(2);
    
    torch::Tensor out = torch::empty({B, D, gates.size(0)}, tokens.options());
    const int batch_stride_o = out.strides()[0];

    dim3 grid(B, D, out.strides()[1]); // 4 is sequential steps;
    // dim3 grid(1, 1, 1); // 4 is sequential steps;

    // DISPATCH_FLOAT_AND_HALF_AND_BF16(tokens.scalar_type(), gates.scalar_type(),
    //     "depthwise conv 1d fwd bhl",
    //     ([&]
    //         { scan<input_t, weight_t, 8, 1><<<grid, seqlen>>>(
    //                 static_cast<weight_t *>(gates.data_ptr()),
    //                 static_cast<input_t *>(tokens.data_ptr()),
    //                 static_cast<input_t *>(out.data_ptr()),
    //                 batch_stride, dim_stride, batch_stride_o
    //             );
    //         }
    //     )
    // ); 
    scan<__half, __half, 8, 1><<<grid, seqlen>>>(
        static_cast<__half *>(gates.data_ptr()),
        static_cast<__half *>(tokens.data_ptr()),
        static_cast<__half *>(out.data_ptr()),
        batch_stride, dim_stride, batch_stride_o
    );
    return out;
}
