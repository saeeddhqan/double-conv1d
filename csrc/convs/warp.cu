#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>

template <typename weight_t, int kNWarpsPerBlock, int kNChunksPerSequence>
__global__ void scan(
    const weight_t* gates,
    const weight_t* tokens,
    weight_t* result,
    const int batch_stride,
    const int dim_stride,
    const bool reverse
) {
    //  256 number of each conv window (after pad), so it isn't a magic number
    // 32 is warp size. lol
    __shared__ weight_t warpLastToken[kNWarpsPerBlock];
    const uint lane_id = threadIdx.x & 31; // x % 32
    // resets after each chunk
    const int warp_id = (threadIdx.x / 32);

    int odd;
    // 8 is the number of warps within each conv window
    int c_out_id = (int)(floor(threadIdx.x / 256) + 1e-8) * 8;
    int offset = (blockIdx.x * batch_stride + blockIdx.y * dim_stride);

    for (int i = 0; i < kNChunksPerSequence; ++i)
    {
        if (i) {
            __syncthreads();
            offset += blockDim.x;
        }


        weight_t x = tokens[offset + threadIdx.x] * gates[threadIdx.x + (i * blockDim.x)];
        odd = lane_id & 1;

        #pragma unroll
        for (int delta = 1; delta < 32; delta *= 2) {
            weight_t prev_x = __shfl_up_sync(0xffffffff, x, delta);
            if (lane_id >= delta && odd) {
                x += prev_x;
                odd = (lane_id % (delta * 4)) == (delta * 4) - 1;
            }
        }
        __syncwarp();
        if (lane_id == 31) {
            warpLastToken[warp_id] = x;
        }
        __syncthreads();
        if (threadIdx.x % 256 == 255) {
            printf("%d, %d, %d, %f, %d\n", threadIdx.x, i, c_out_id, (float)x, offset);
            x += warpLastToken[0 + c_out_id];
            x += warpLastToken[1 + c_out_id];
            x += warpLastToken[2 + c_out_id];
            x += warpLastToken[3 + c_out_id];
            x += warpLastToken[4 + c_out_id];
            x += warpLastToken[5 + c_out_id];
            x += warpLastToken[6 + c_out_id];
            result[offset + threadIdx.x] = x;
        }
    }
}

template <typename weight_t, typename torch_weight_t>
void
warpscan(const at::Tensor &gates, const at::Tensor &tokens, const at::Tensor &out, const bool reverse) {
    const auto strides = tokens.strides();
    const int batch_stride = strides[0];
    const int dim_stride = strides[1];
    const int gate_stride = gates.size(0) / 8;
    TORCH_CHECK(tokens.stride(-1) == 1 || tokens.size(-1) == 1);
    TORCH_CHECK(gates.stride(-1) == 1 || gates.size(-1) == 1);

    const auto sizes = tokens.sizes();
    const int batch_size = sizes[0];
    const int dim = sizes[1];
    const int seqlen = sizes[2];

    auto stream = at::cuda::getCurrentCUDAStream().stream();
    dim3 grid(batch_size, dim);

    if (seqlen == 256) {
        constexpr int kNWarpsPerBlock = 8;
        constexpr int kNChunksPerSequence = 1;
        scan<weight_t, kNWarpsPerBlock, kNChunksPerSequence><<<grid, seqlen, kNWarpsPerBlock * sizeof(weight_t), stream>>>(
            reinterpret_cast<weight_t*>(gates.data_ptr<torch_weight_t>()), reinterpret_cast<weight_t*>(tokens.data_ptr<torch_weight_t>()), reinterpret_cast<weight_t*>(out.data_ptr<torch_weight_t>()),
            batch_stride, dim_stride, reverse
        );
    } else if (seqlen == 512) {
        constexpr int kNWarpsPerBlock = 16;
        constexpr int kNChunksPerSequence = 1;
        scan<weight_t, kNWarpsPerBlock, kNChunksPerSequence><<<grid, seqlen, kNWarpsPerBlock * sizeof(weight_t), stream>>>(
            reinterpret_cast<weight_t*>(gates.data_ptr<torch_weight_t>()), reinterpret_cast<weight_t*>(tokens.data_ptr<torch_weight_t>()), reinterpret_cast<weight_t*>(out.data_ptr<torch_weight_t>()),
            batch_stride, dim_stride, reverse
        );
    } else if (seqlen == 1024) {
        constexpr int kNWarpsPerBlock = 16;
        constexpr int kNChunksPerSequence = 2;
        scan<weight_t, kNWarpsPerBlock, kNChunksPerSequence><<<grid, 512, kNWarpsPerBlock * sizeof(weight_t), stream>>>(
            reinterpret_cast<weight_t*>(gates.data_ptr<torch_weight_t>()), reinterpret_cast<weight_t*>(tokens.data_ptr<torch_weight_t>()), reinterpret_cast<weight_t*>(out.data_ptr<torch_weight_t>()),
            batch_stride, dim_stride, reverse
        );
    } else if (seqlen == 2048) {
        constexpr int kNWarpsPerBlock = 16;
        constexpr int kNChunksPerSequence = 4;
        scan<weight_t, kNWarpsPerBlock, kNChunksPerSequence><<<grid, 512, kNWarpsPerBlock * sizeof(weight_t), stream>>>(
            reinterpret_cast<weight_t*>(gates.data_ptr<torch_weight_t>()), reinterpret_cast<weight_t*>(tokens.data_ptr<torch_weight_t>()), reinterpret_cast<weight_t*>(out.data_ptr<torch_weight_t>()),
            batch_stride, dim_stride, reverse
        );
    } else if (seqlen == 4096) {
        constexpr int kNWarpsPerBlock = 16;
        constexpr int kNChunksPerSequence = 8;
        scan<weight_t, kNWarpsPerBlock, kNChunksPerSequence><<<grid, 512, kNWarpsPerBlock * sizeof(weight_t), stream>>>(
            reinterpret_cast<weight_t*>(gates.data_ptr<torch_weight_t>()), reinterpret_cast<weight_t*>(tokens.data_ptr<torch_weight_t>()), reinterpret_cast<weight_t*>(out.data_ptr<torch_weight_t>()),
            batch_stride, dim_stride, reverse
        );
    } else if (seqlen == 8192) {
        constexpr int kNWarpsPerBlock = 16;
        constexpr int kNChunksPerSequence = 16;
        scan<weight_t, kNWarpsPerBlock, kNChunksPerSequence><<<grid, 512, kNWarpsPerBlock * sizeof(weight_t), stream>>>(
            reinterpret_cast<weight_t*>(gates.data_ptr<torch_weight_t>()), reinterpret_cast<weight_t*>(tokens.data_ptr<torch_weight_t>()), reinterpret_cast<weight_t*>(out.data_ptr<torch_weight_t>()),
            batch_stride, dim_stride, reverse
        );
    } else if (seqlen == 16384) {
        constexpr int kNWarpsPerBlock = 16;
        constexpr int kNChunksPerSequence = 32;
        scan<weight_t, kNWarpsPerBlock, kNChunksPerSequence><<<grid, 512, kNWarpsPerBlock * sizeof(weight_t), stream>>>(
            reinterpret_cast<weight_t*>(gates.data_ptr<torch_weight_t>()), reinterpret_cast<weight_t*>(tokens.data_ptr<torch_weight_t>()), reinterpret_cast<weight_t*>(out.data_ptr<torch_weight_t>()),
            batch_stride, dim_stride, reverse
        );
    } else {
        TORCH_CHECK(false && "seqlen must be a power of 2, >= 32, <= 65536");
    }
}

at::Tensor
warpscan_forward(const at::Tensor &gates, const at::Tensor &tokens, const at::Tensor &out, const bool reverse) {
    TORCH_CHECK(tokens.is_cuda());
    TORCH_CHECK(gates.is_cuda());
    TORCH_CHECK(tokens.is_contiguous());
    TORCH_CHECK(gates.is_contiguous());

    if (tokens.scalar_type() == at::ScalarType::BFloat16) {
        TORCH_CHECK(gates.scalar_type() == at::ScalarType::BFloat16);
        warpscan<__hip_bfloat16, at::BFloat16>(gates, tokens, out, reverse);
    } else if (tokens.scalar_type() == at::ScalarType::Half) {
        TORCH_CHECK(gates.scalar_type() == at::ScalarType::Half);
        warpscan<__half, at::Half>(gates, tokens, out, reverse);
    } else if (tokens.scalar_type() == at::ScalarType::Float) {
        TORCH_CHECK(gates.scalar_type() == at::ScalarType::Float);
        warpscan<float, float>(gates, tokens, out, reverse);
    } else {
        TORCH_CHECK(false && "Unsupported tensor dtype: expecting bfloat16, float16 or float32");
    }
    return out;
}